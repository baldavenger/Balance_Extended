
#include <hip/hip_runtime.h>
const float PIE = 3.14159265358979323846264338327950288f;
const float X_BRK = 0.0078125f;
const float Y_BRK = 0.155251141552511f;
const float A = 10.5402377416545f;
const float B = 0.0729055341958355f;
//float3 RGB_2_YAB_MAT[3] = { {1.0f/3.0f, 1.0f/2.0f, 0.0f},
//								  {1.0f/3.0f, -1.0f/4.0f, 0.433012701892219f}, 
//								  {1.0f/3.0f, -1.0f/4.0f, -0.433012701892219f} };

__device__ float clamp(float in, float low, float high)
{
float out;
out = in < low ? low : in > high ? high : in;
return out;
}

__device__ float3* mult_f_f33(float f, float3 A[3])
{
for( int i = 0; i < 3; ++i )
{
A[i].x *= f;
A[i].y *= f;
A[i].z *= f;
}
return A;
}

__device__ float3 mult_f3_f33(float3 In, const float3 A[3])
{
float3 out;
out.x = In.x * A[0].x + In.y * A[0].y + In.z * A[0].z;
out.y = In.x * A[1].x + In.y * A[1].y + In.z * A[1].z;
out.z = In.x * A[2].x + In.y * A[2].y + In.z * A[2].z;
return out;
}

__device__ float3* invert_f33(float3 A[3])
{
float3 result[3];
float det =   A[0].x * A[1].y * A[2].z
			+ A[0].y * A[1].z * A[2].x
			+ A[0].z * A[1].x * A[2].y
			- A[2].x * A[1].y * A[0].z
			- A[2].y * A[1].z * A[0].x
			- A[2].z * A[1].x * A[0].y;
			
if( det != 0.0f )
{
result[0].x = A[1].y * A[2].z - A[1].z * A[2].y;
result[0].y = A[2].y * A[0].z - A[2].z * A[0].y;
result[0].z = A[0].y * A[1].z - A[0].z * A[1].y;
result[1].x = A[2].x * A[1].z - A[1].x * A[2].z;
result[1].y = A[0].x * A[2].z - A[2].x * A[0].z;
result[1].z = A[1].x * A[0].z - A[0].x * A[1].z;
result[2].x = A[1].x * A[2].y - A[2].x * A[1].y;
result[2].y = A[2].x * A[0].y - A[0].x * A[2].y;
result[2].z = A[0].x * A[1].y - A[1].x * A[0].y;

A =  mult_f_f33( 1.0f / det, result);
}
return A;
}

__device__ float3 rgb_2_yab(float3 rgb)
{
float3 RGB_2_YAB_MAT[3] = { {1.0f/3.0f, 1.0f/2.0f, 0.0f},
							{1.0f/3.0f, -1.0f/4.0f, 0.433012701892219f}, 
							{1.0f/3.0f, -1.0f/4.0f, -0.433012701892219f} };
float3 yab;
yab = mult_f3_f33(rgb, RGB_2_YAB_MAT);
return yab;
}

__device__ float3 yab_2_ych(float3 yab)
{
float3 ych;
ych = yab;
float yo = yab.y * yab.y + yab.z * yab.z;
ych.y = sqrtf(yo);
ych.z = atan2f(yab.z, yab.y) * (180.0f / PIE);
if (ych.z < 0.0f)
{
ych.z += 360.0f;
}
return ych;
}

__device__ float3 ych_2_yab(float3 ych) 
{
float3 yab;
yab.x = ych.x;
float h = ych.z * (PIE / 180.0f);
yab.y = ych.y * cosf(h);
yab.z = ych.y * sinf(h);
return yab;
}

__device__ float3 yab_2_rgb(float3 yab)
{
float3 RGB_2_YAB_MAT[3] = { {1.0f/3.0f, 1.0f/2.0f, 0.0f},
							{1.0f/3.0f, -1.0f/4.0f, 0.433012701892219f}, 
							{1.0f/3.0f, -1.0f/4.0f, -0.433012701892219f} };
float3 rgb;
float3* abc; 
abc = invert_f33(RGB_2_YAB_MAT);
rgb = mult_f3_f33(yab, abc);
return rgb;
}

__device__ float3 scale_C(float3 rgb, float percentC)
{
float3 ych, yab;
yab = rgb_2_yab(rgb);
ych = yab_2_ych(yab);
ych.y *= percentC;
yab = ych_2_yab(ych);
rgb = yab_2_rgb(yab);
return rgb;
}

__device__ float lin_to_ACEScct(float in)
{
float out;
if (in <= X_BRK){
out = A * in + B;
} else {
out = (log2f(in) + 9.72f) / 17.52f;
}
return out;
}

__device__ float ACEScct_to_lin(float in)
{
float out;    
if (in > Y_BRK){
out = powf(2.0f, in * 17.52f - 9.72f);
} else {
out = (in - B) / A;
}
return out;
}

__device__ float3 ACES_to_ACEScct(float3 in)
{
float3 out;

//AP0 to AP1
out.x =  1.4514393161f * in.x + -0.2365107469f * in.y + -0.2149285693f * in.z;
out.y = -0.0765537734f * in.x +  1.1762296998f * in.y + -0.0996759264f * in.z;
out.z =  0.0083161484f * in.x + -0.0060324498f * in.y +  0.9977163014f * in.z;

// Linear to ACEScct
out.x = lin_to_ACEScct(out.x);
out.y = lin_to_ACEScct(out.y);
out.z = lin_to_ACEScct(out.z);

return out;
}

__device__ float3 ACEScct_to_ACES(float3 in)
{
float3 lin, out;

// ACEScct to linear
lin.x = ACEScct_to_lin(in.x);
lin.y = ACEScct_to_lin(in.y);
lin.z = ACEScct_to_lin(in.z);

// AP1 to AP0
out.x =  0.6954522414f * lin.x +  0.1406786965f * lin.y +  0.1638690622f * lin.z;
out.y =  0.0447945634f * lin.x +  0.8596711185f * lin.y +  0.0955343182f * lin.z;
out.z = -0.0055258826f * lin.x +  0.0040252103f * lin.y +  1.0015006723f * lin.z;

return out;
}

__device__ float3 ASCCDL_inACEScct
(
float3 acesIn, 
float SLOPE[3],
float OFFSET[3],
float POWER[3],
float SAT
)
{

acesIn = ACES_to_ACEScct(acesIn);

//acesIn.x = powf(clamp((acesIn.x * SLOPE[0]) + OFFSET[0], 0.0f, 1.0f), POWER[0]);
//acesIn.y = powf(clamp((acesIn.y * SLOPE[1]) + OFFSET[1], 0.0f, 1.0f), POWER[1]);
//acesIn.z = powf(clamp((acesIn.z * SLOPE[2]) + OFFSET[2], 0.0f, 1.0f), POWER[2]);

float sopR = clamp((acesIn.x * SLOPE[0]) + OFFSET[0], 0.0f, 1.0f);
float sopG = clamp((acesIn.y * SLOPE[1]) + OFFSET[1], 0.0f, 1.0f);
float sopB = clamp((acesIn.z * SLOPE[2]) + OFFSET[2], 0.0f, 1.0f);

acesIn.x = powf(sopR, POWER[0]);
acesIn.y = powf(sopG, POWER[1]);
acesIn.z = powf(sopB, POWER[2]);

float luma = 0.2126f *acesIn.x + 0.7152f * acesIn.y + 0.0722f * acesIn.z;

float satClamp = clamp(SAT, 0.0f, 10.0f);    
acesIn.x = luma + satClamp * (acesIn.x - luma);
acesIn.y = luma + satClamp * (acesIn.y - luma);
acesIn.z = luma + satClamp * (acesIn.z - luma);

acesIn = ACEScct_to_ACES(acesIn);

return acesIn;
}

__device__ float3 gamma_adjust_linear(float3 rgbIn, float GAMMA, float PIVOT)
{
const float SCALAR = PIVOT / powf(PIVOT, GAMMA);

if (rgbIn.x > 0.0f){ rgbIn.x = powf(rgbIn.x, GAMMA) * SCALAR;}
if (rgbIn.y > 0.0f){ rgbIn.y = powf(rgbIn.y, GAMMA) * SCALAR;}
if (rgbIn.z > 0.0f){ rgbIn.z = powf(rgbIn.z, GAMMA) * SCALAR;}
return rgbIn;
}

__device__ float interpolate1D(float2 table[], float p, int t)
{
if( p <= table[0].x ) return table[0].y;
if( p >= table[t - 1].x ) return table[t - 1].y;

for( int i = 0; i < t - 1; ++i )
{
if( table[i].x <= p && p < table[i+1].x )
{
float s = (p - table[i].x) / (table[i+1].x - table[i].x);
return table[i].y * ( 1.0f - s ) + table[i+1].y * s;
}
}
return 0.0f;
}

__device__ float cubic_basis_shaper(float x, float w)
{
  float4 M[4] = { { -1./6,  3./6, -3./6,  1./6 },
                {  3./6, -6./6,  3./6,  0./6 },
                { -3./6,  0./6,  3./6,  0./6 },
                {  1./6,  4./6,  1./6,  0./6 } };
  
float knots[5] = { -w/2.0f, -w/4.0f, 0.0f, w/4.0f, w/2.0f };

float y = 0.0f;
if ((x > knots[0]) && (x < knots[4])) {  
float knot_coord = (x - knots[0]) * 4.0f/w;  
int j = knot_coord;
float t = knot_coord - j;

float monomials[4] = { t*t*t, t*t, t, 1. };

if ( j == 3) {
y = monomials[0] * M[0].x + monomials[1] * M[1].x + 
	monomials[2] * M[2].x + monomials[3] * M[3].x;
} else if ( j == 2) {
y = monomials[0] * M[0].y + monomials[1] * M[1].y + 
	monomials[2] * M[2].y + monomials[3] * M[3].y;
} else if ( j == 1) {
y = monomials[0] * M[0].z + monomials[1] * M[1].z + 
	monomials[2] * M[2].z + monomials[3] * M[3].z;
} else if ( j == 0) {
y = monomials[0] * M[0].w + monomials[1] * M[1].w + 
	monomials[2] * M[2].w + monomials[3] * M[3].w;
} else {
y = 0.0f;
}
}

return y * 3/2.0f;
}

__device__ float center_hue( float hue, float centerH)
{
float hueCentered = hue - centerH;
if (hueCentered < -180.0f) hueCentered = hueCentered + 360.0f;
else if (hueCentered > 180.0f) hueCentered = hueCentered - 360.0f;
return hueCentered;
}

__device__ float uncenter_hue( float hueCentered, float centerH)
{
float hue = hueCentered + centerH;
if (hue < 0.0f) hue = hue + 360.0f;
else if (hue > 360.0f) hue = hue - 360.0f;
return hue;
}

__device__ float3 rotate_H_in_H(float3 rgb, float centerH, float widthH, float degreesShift)
{
float3 ych, yab;
yab = rgb_2_yab(rgb);
ych = yab_2_ych(yab);

float centeredHue = center_hue(ych.z, centerH);
float f_H = cubic_basis_shaper(centeredHue, widthH);

float old_hue = centeredHue;
float new_hue = centeredHue + degreesShift;
float2 table[2] = {{0.0f, old_hue}, {1.0f, new_hue}};
float blended_hue = interpolate1D(table, f_H, 2);
 
if (f_H > 0.0f)
{
ych.z = uncenter_hue(blended_hue, centerH);
}

yab = ych_2_yab(ych);
rgb = yab_2_rgb(yab);
return rgb;
}

__device__ float3 scale_C_at_H
( 
float3 rgb, 
float centerH,
float widthH,
float percentC
)
{
float3 ych, yab, new_rgb;
new_rgb = rgb;
yab = rgb_2_yab(rgb);
ych = yab_2_ych(yab);
if (ych.y > 0.0f) {
float centeredHue = center_hue(ych.z, centerH);
float f_H = cubic_basis_shaper(centeredHue, widthH);
if (f_H > 0.0) {
float3 new_ych = ych;
new_ych.y = ych.y * (f_H * (percentC - 1.0f) + 1.0f);
yab = ych_2_yab(new_ych);
new_rgb = yab_2_rgb(yab);
} else { 
new_rgb = rgb; 
}
}
return new_rgb;
}

__device__ float3 transform(int p_Width, int p_Height, int p_X, int p_Y, float p_R, float p_G, float p_B)
{
float3 Aces;
Aces.x = p_R;
Aces.y = p_G;
Aces.z = p_B;

Aces = scale_C(Aces, 0.7f);

float SLOPE[3] = {1.0f, 1.0f, 0.94f};
float OFFSET[3] = {0.0f, 0.0f, 0.02f};
float POWER[3] = {1.0f, 1.0f, 1.0f};
float SAT = 1.0f;

Aces = ASCCDL_inACEScct(Aces, SLOPE, OFFSET, POWER, SAT);

Aces = gamma_adjust_linear(Aces, 1.5f, 0.18f);

Aces = rotate_H_in_H(Aces, 0.0f, 30.0f, 5.0f);

Aces = rotate_H_in_H(Aces, 80.0f, 60.0f, -15.0f);

Aces = rotate_H_in_H(Aces, 52.0f, 50.0f, -14.0f);

Aces = scale_C_at_H(Aces, 45.0f, 40.0f, 1.4f);

Aces = rotate_H_in_H(Aces, 190.0f, 40.0f, 30.0f);

Aces = scale_C_at_H(Aces, 240.0f, 120.0f, 1.4f);

return Aces;
}

__global__ void LMTKernel(const float* p_Input, float* p_Output, int p_Width, int p_Height, float p_Scale1, float p_Scale2, 
float p_Scale3, float p_Scale4, float p_Scale5, float p_Scale6, float p_Scale7, float p_Scale8, float p_Scale9, float p_Scale10, 
float p_Scale11, float p_Scale12, float p_Scale13, float p_Scale14, float p_Scale15, float p_Scale16, float p_Scale17, float p_Scale18, 
float p_Scale19, float p_Scale20, float p_Scale21, float p_Scale22, float p_Scale23, float p_Scale24, float p_Scale25, float p_Scale26, 
float p_Scale27, float p_Scale28, float p_Scale29, float p_Scale30, float p_Scale31)
{
   const int x = blockIdx.x * blockDim.x + threadIdx.x;
   const int y = blockIdx.y * blockDim.y + threadIdx.y;
   
	
   if ((x < p_Width) && (y < p_Height))
   {
	const int index = ((y * p_Width) + x) * 4;
	
	float3 Aces;
	Aces.x = p_Input[index + 0];
	Aces.y = p_Input[index + 1];
	Aces.z = p_Input[index + 2];
	
	Aces = scale_C(Aces, p_Scale1);

	float SLOPE[3] = {p_Scale2, p_Scale3, p_Scale4};
	float OFFSET[3] = {p_Scale5, p_Scale6, p_Scale7};
	float POWER[3] = {p_Scale8, p_Scale9, p_Scale10};
	float SAT = p_Scale11;

	Aces = ASCCDL_inACEScct(Aces, SLOPE, OFFSET, POWER, SAT);

	Aces = gamma_adjust_linear(Aces, p_Scale12, p_Scale13);

	Aces = rotate_H_in_H(Aces, p_Scale14, p_Scale15, p_Scale16);

	Aces = rotate_H_in_H(Aces, p_Scale17, p_Scale18, p_Scale19);

	Aces = rotate_H_in_H(Aces, p_Scale20, p_Scale21, p_Scale22);

	Aces = scale_C_at_H(Aces, p_Scale23, p_Scale24, p_Scale25);

	Aces = rotate_H_in_H(Aces, p_Scale26, p_Scale27, p_Scale28);

	Aces = scale_C_at_H(Aces, p_Scale29, p_Scale30, p_Scale31);
																												   
	p_Output[index + 0] = Aces.x;
	p_Output[index + 1] = Aces.y;
	p_Output[index + 2] = Aces.z;
	p_Output[index + 3] = p_Input[index + 3];
   }
}

void RunCudaKernel(const float* p_Input, float* p_Output, int p_Width, int p_Height, float* p_Scale)
{
    dim3 threads(128, 1, 1);
    dim3 blocks(((p_Width + threads.x - 1) / threads.x), p_Height, 1);

    LMTKernel<<<blocks, threads>>>(p_Input, p_Output, p_Width, p_Height, p_Scale[0], p_Scale[1], 
    p_Scale[2], p_Scale[3], p_Scale[4], p_Scale[5], p_Scale[6], p_Scale[7], p_Scale[8], p_Scale[9], p_Scale[10], 
    p_Scale[11], p_Scale[12], p_Scale[13], p_Scale[14], p_Scale[15], p_Scale[16], p_Scale[17], p_Scale[18], p_Scale[19], 
    p_Scale[20], p_Scale[21], p_Scale[22], p_Scale[23], p_Scale[24], p_Scale[25], p_Scale[26], p_Scale[27], p_Scale[28], 
    p_Scale[29], p_Scale[30]);
}
